/* Copyright 2023 The GPU4PySCF Authors. All Rights Reserved.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS        32

__global__
static void _calc_distances(double *dist, const double *x, const double *y, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n || j >= n){
        return;
    }

    double dx = x[3*i]   - y[3*j];
    double dy = x[3*i+1] - y[3*j+1];
    double dz = x[3*i+2] - y[3*j+2];
    dist[i*n+j] = norm3d(dx, dy, dz);
}

extern "C" {
int dist_matrix(hipStream_t stream, double *dist, const double *x, const double *y, int n)
{
    int ntile = (n + THREADS - 1) / THREADS;
    dim3 threads(THREADS, THREADS);
    dim3 blocks(ntile, ntile);
    _calc_distances<<<blocks, threads, 0, stream>>>(dist, x, y, n);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return 1;
    }
    return 0;
}
}
